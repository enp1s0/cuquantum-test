#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <custatevec.h>
#include <iostream>

using statevector_t = hipDoubleComplex;

template <class T>
struct get_real;
template <> struct get_real<hipDoubleComplex> {using type = double;};
template <> struct get_real<hipComplex> {using type = float;};

template <class T>
hipDataType get_data_type();
template <> hipDataType get_data_type<hipComplex      >() {return HIP_C_32F;}
template <> hipDataType get_data_type<hipDoubleComplex>() {return HIP_C_64F;}

template <class T>
custatevecComputeType_t get_custatevec_compute_type();
template <> custatevecComputeType_t  get_custatevec_compute_type<hipComplex      >() {return CUSTATEVEC_COMPUTE_32F;}
template <> custatevecComputeType_t  get_custatevec_compute_type<hipDoubleComplex>() {return CUSTATEVEC_COMPUTE_64F;}

void check_custatevec_error(
		const custatevecStatus_t status,
		const char* const filepath,
		const unsigned line,
		const char* const funcname
		) {
	if (status != CUSTATEVEC_STATUS_SUCCESS) {
		auto get_str = [](const custatevecStatus_t state) {
			switch(state) {
#define CUSTATEVEC_ERROR_CASE_MESSAGE(x) case x: return #x
				CUSTATEVEC_ERROR_CASE_MESSAGE(CUSTATEVEC_STATUS_SUCCESS                 );
				CUSTATEVEC_ERROR_CASE_MESSAGE(CUSTATEVEC_STATUS_NOT_INITIALIZED         );
				CUSTATEVEC_ERROR_CASE_MESSAGE(CUSTATEVEC_STATUS_ALLOC_FAILED            );
				CUSTATEVEC_ERROR_CASE_MESSAGE(CUSTATEVEC_STATUS_INVALID_VALUE           );
				CUSTATEVEC_ERROR_CASE_MESSAGE(CUSTATEVEC_STATUS_ARCH_MISMATCH           );
				CUSTATEVEC_ERROR_CASE_MESSAGE(CUSTATEVEC_STATUS_EXECUTION_FAILED        );
				CUSTATEVEC_ERROR_CASE_MESSAGE(CUSTATEVEC_STATUS_INTERNAL_ERROR          );
				CUSTATEVEC_ERROR_CASE_MESSAGE(CUSTATEVEC_STATUS_NOT_SUPPORTED           );
				CUSTATEVEC_ERROR_CASE_MESSAGE(CUSTATEVEC_STATUS_INSUFFICIENT_WORKSPACE  );
				CUSTATEVEC_ERROR_CASE_MESSAGE(CUSTATEVEC_STATUS_SAMPLER_NOT_PREPROCESSED);
			default:
				return "Unknown";
			}
		};
		std::printf("ERROR: %s @ %s, line %u (%s)\n", get_str(status), filepath, line, funcname);
		exit(1);
	}
}

#define CHECH_CUSTATEVEC_ERROR(status) check_custatevec_error((status), __FILE__, __LINE__, __func__)

void init_statevector(statevector_t* const ptr,
		const std::size_t statevector_length) {
	hipMemset(ptr, 0, sizeof(statevector_t) * statevector_length / sizeof(int));

	statevector_t zero;
	zero.x = 1;
	zero.y = 0;

	hipMemcpy(ptr, &zero, sizeof(zero), hipMemcpyDefault);
}

void gate_H(custatevecHandle_t handle,
		statevector_t* const ptr,
		const unsigned target_qubit,
		const unsigned num_qubits) {
	const auto sqrt2 = std::sqrt(2.);
	constexpr unsigned adjoint = 0;
	statevector_t matrix[4];
	for (unsigned i = 0; i < 4; i++) {
		matrix[i].x = 1 / sqrt2;
		matrix[i].y = 1 / sqrt2;
	}
	matrix[3].x *= -1;
	matrix[3].y *= -1;

	std::printf("[%10s] custatevecApplyMatrix_bufferSize start\n", __func__);
	void* working_memory;
	std::size_t working_memory_size;
	CHECH_CUSTATEVEC_ERROR(custatevecApplyMatrix_bufferSize(
			handle,
			get_data_type<statevector_t>(),
			num_qubits,
			matrix,
			get_data_type<statevector_t>(),
			CUSTATEVEC_MATRIX_LAYOUT_COL,
			adjoint,
			1,
			0,
			get_custatevec_compute_type<statevector_t>(),
			&working_memory_size
			));

	if (working_memory_size) {
		hipMalloc(&working_memory, working_memory_size);
	}
	std::printf("[%10s] hipMalloc start (%lu B)\n", __func__, working_memory_size);

	std::printf("[%10s] custatevecApplyMatrix start\n", __func__);
	int targets[] = {(int)target_qubit};
	int controls[] = {};
	CHECH_CUSTATEVEC_ERROR(custatevecApplyMatrix(
			handle,
			ptr,
			get_data_type<statevector_t>(),
			num_qubits,
			matrix,
			get_data_type<statevector_t>(),
			CUSTATEVEC_MATRIX_LAYOUT_COL,
			adjoint,
			targets,
			1,
			controls,
			0,
			nullptr,
			get_custatevec_compute_type<statevector_t>(),
			working_memory,
			working_memory_size
			));

	hipFree(working_memory);
}

int main() {
	constexpr unsigned num_qubits = 29;
	constexpr std::size_t statevector_length = 1lu << num_qubits;

	statevector_t *statevector;
	hipMalloc(&statevector, sizeof(statevector_t) * statevector_length);

	init_statevector(statevector, statevector_length);

	std::printf("[%10s] custatevecCreate start\n", __func__);
	custatevecHandle_t handle;
	CHECH_CUSTATEVEC_ERROR(custatevecCreate(&handle));

	for (unsigned i = 0; i < num_qubits; i++) {
		gate_H(handle, statevector, i, num_qubits);
	}

	std::printf("[%10s] custatevecDestroy start\n", __func__);
	CHECH_CUSTATEVEC_ERROR(custatevecDestroy(handle));

	hipFree(statevector);

	return 0;
}
